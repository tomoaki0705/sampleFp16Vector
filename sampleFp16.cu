#include "hip/hip_runtime.h"

// clamp x to range [a, b]
__device__ unsigned char clamp(float x, float a, float b)
{
    return (unsigned char)(max(a, min(b, x)));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__global__ void
cudaProcessHalf(unsigned char *g_odata, short *g_indata, unsigned char* imageData, int imgw)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	unsigned short a = g_indata[y*imgw+x];
	float gain;
	gain = __half2float(a);

	float b = imageData[(y*imgw+x)*3  ];
	float g = imageData[(y*imgw+x)*3+1];
	float r = imageData[(y*imgw+x)*3+2];

	g_odata[(y*imgw+x)*3  ] = clamp(b * gain, 0.0f, 255.0f);
	g_odata[(y*imgw+x)*3+1] = clamp(g * gain, 0.0f, 255.0f);
	g_odata[(y*imgw+x)*3+2] = clamp(r * gain, 0.0f, 255.0f);
}

extern "C" void
launchCudaProcessHalf(dim3 grid, dim3 block, int sbytes,
						short *gain,
						unsigned char *imageInput,
						unsigned char *imageOutput,
						int imgw)
{
    cudaProcessHalf<<< grid, block, sbytes >>>(imageOutput, gain, imageInput, imgw);

}

__global__ void
cudaProcessFloat(unsigned char *g_odata, float *g_indata, unsigned char* imageData, int imgw)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	float gain = g_indata[y*imgw+x];

	float b = imageData[(y*imgw+x)*3  ];
	float g = imageData[(y*imgw+x)*3+1];
	float r = imageData[(y*imgw+x)*3+2];

	g_odata[(y*imgw+x)*3  ] = clamp(b * gain, 0.0f, 255.0f);
	g_odata[(y*imgw+x)*3+1] = clamp(g * gain, 0.0f, 255.0f);
	g_odata[(y*imgw+x)*3+2] = clamp(r * gain, 0.0f, 255.0f);
}

extern "C" void
launchCudaProcessFloat(dim3 grid, dim3 block, int sbytes,
						float *gain,
						unsigned char *imageInput,
						unsigned char *imageOutput,
						int imgw)
{
    cudaProcessFloat<<< grid, block, sbytes >>>(imageOutput, gain, imageInput, imgw);

}
