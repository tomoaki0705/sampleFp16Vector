#include "hip/hip_runtime.h"

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__global__ void
cudaProcessHalf(unsigned int *g_odata, short *g_indata, unsigned int* imageData, int imgw)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	unsigned short a = g_indata[y*imgw+x];
	float gain;
	gain = __half2float(a);

	unsigned int p = imageData[y*imgw+x];

	float b = (float)((p >> 16) & 0xff);
	float g = (float)((p >>  8) & 0xff);
	float r = (float)((p      ) & 0xff);

	uchar4 c4;
	c4.x = (unsigned char)(b * gain);
	c4.y = (unsigned char)(g * gain);
	c4.z = (unsigned char)(r * gain);
	g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void
launchCudaProcessHalf(dim3 grid, dim3 block, int sbytes,
						short *gain,
						unsigned int *imageInput,
						unsigned int *imageOutput,
						int imgw)
{
    cudaProcessHalf<<< grid, block, sbytes >>>(imageOutput, gain, imageInput, imgw);

}

__global__ void
cudaProcessFloat(unsigned int *g_odata, float *g_indata, unsigned int* imageData, int imgw)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x*bw + tx;
	int y = blockIdx.y*bh + ty;

	float gain = g_indata[y*imgw+x];

	unsigned int p = imageData[y*imgw+x];

	float b = (float)((p >> 16) & 0xff);
	float g = (float)((p >>  8) & 0xff);
	float r = (float)((p      ) & 0xff);

	uchar4 c4;
	c4.x = (unsigned char)(b * gain);
	c4.y = (unsigned char)(g * gain);
	c4.z = (unsigned char)(r * gain);
	g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void
launchCudaProcessFloat(dim3 grid, dim3 block, int sbytes,
						float *gain,
						unsigned int *imageInput,
						unsigned int *imageOutput,
						int imgw)
{
    cudaProcessFloat<<< grid, block, sbytes >>>(imageOutput, gain, imageInput, imgw);

}
